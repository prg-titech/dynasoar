#include "hip/hip_runtime.h"
#include <chrono>

#include "generation.h"
#include "../configuration.h"
#include "../dataset_loader.h"
#include "../rendering.h"


// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;


#ifdef OPTION_RENDER
// Rendering array.
// TODO: Fix variable names.
__device__ int* device_render_cells;
int* host_render_cells;
int* d_device_render_cells;
#endif  // OPTION_RENDER


// Dataset.
__device__ int SIZE_X;
__device__ int SIZE_Y;
__device__ Cell** cells;
dataset_t dataset;


__device__ Cell::Cell() : agent_(nullptr), reserved_(0) {}


__device__ Agent* Cell::agent() { return agent_; }


__device__ bool Cell::is_empty() { return agent_ == nullptr; }


__device__ Agent::Agent(int cell_id)
    : cell_id_(cell_id), action_(kActionNone) {}


__device__ int Agent::cell_id() { return cell_id_; }


// Only count alive agents in state 0.
__device__ int Agent::num_alive_neighbors() {
  int cell_x = cell_id_ % SIZE_X;
  int cell_y = cell_id_ / SIZE_X;
  int result = 0;

  for (int dx = -1; dx < 2; ++dx) {
    for (int dy = -1; dy < 2; ++dy) {
      int nx = cell_x + dx;
      int ny = cell_y + dy;

      if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
        Alive* alive = cells[ny*SIZE_X + nx]->agent()->cast<Alive>();
        if (alive != nullptr && alive->state() == 0) {
          result++;
        }
      }
    }
  }

  return result;
}


__device__ Alive::Alive(int cell_id)
    : Agent(cell_id), is_new_(true), state_(0) {}


__device__ void Alive::prepare() {
  // If state_ > 0, state simply advance until dead.
  if (state_ == 0) {
    is_new_ = false;

    // Also counts this object itself.
    int alive_neighbors = num_alive_neighbors() - 1;

    const bool stay_alive_param[9] = kStayAlive;
    if (!stay_alive_param[alive_neighbors]) { action_ = kActionDie; }
  }
}


__device__ void Alive::update() {
  int cid = cell_id_;

  // TODO: Consider splitting in two classes for less divergence.
  if (is_new_) {
    // Create candidates in neighborhood.
    create_candidates();
  } else {
    if (action_ == kActionDie && state_ == 0) {
      // Increment state. If reached max. state, replace with Candidate.
      ++state_;
      action_ = kActionNone;
    } else if (state_ > 0 && state_ < kNumStates) {
      ++state_;
    } else if (state_ == kNumStates) {
      // Replace with Candidate.
      cells[cid]->agent_ = new(device_allocator) Candidate(cid);
      destroy(device_allocator, this);
    } else if (state_ > kNumStates) {
      assert(false);
    }
  }
}


__device__ void Alive::create_candidates() {
  assert(is_new_);

  // TODO: Consolidate with Agent::num_alive_neighbors().
  int cell_x = cell_id_ % SIZE_X;
  int cell_y = cell_id_ / SIZE_X;

  for (int dx = -1; dx < 2; ++dx) {
    for (int dy = -1; dy < 2; ++dy) {
      int nx = cell_x + dx;
      int ny = cell_y + dy;

      if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
        auto cid = ny*SIZE_X + nx;
        if (cells[cid]->is_empty()) {
            if (atomicCAS(&cells[cid]->reserved_, 0, 1) == 0) {
              cells[cid]->agent_ = new(device_allocator) Candidate(cid);
            }
        }
      }
    }
  }
}


__device__ Candidate::Candidate(int cell_id) : Agent(cell_id) {}


__device__ void Candidate::prepare() {
  int alive_neighbors = num_alive_neighbors();
  const bool spawn_param[9] = kSpawnNew;

  if (spawn_param[alive_neighbors]) {
    action_ = kActionSpawnAlive;
  } else if (alive_neighbors == 0) {
    action_ = kActionDie;
  }
}


__device__ void Candidate::update() {
  // TODO: Why is this necessary?
  int cid = cell_id_;

  if (action_ == kActionSpawnAlive) {
    cells[cid]->agent_ = new(device_allocator) Alive(cid);
    destroy(device_allocator, this);
  } else if (action_ == kActionDie) {
    cells[cid]->agent_ = nullptr;
    cells[cid]->reserved_ = 0;
    destroy(device_allocator, this);
  }
}


__global__ void create_cells() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    cells[i] = new(device_allocator) Cell();
  }
}


// Must be followed by Alive::update().
__global__ void load_game(int* cell_ids, int num_cells) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_cells; i += blockDim.x * gridDim.x) {
    cells[cell_ids[i]]->agent_ = new(device_allocator) Alive(cell_ids[i]);
    assert(cells[cell_ids[i]]->agent()->cell_id() == cell_ids[i]);
  }
}


__global__ void initialize_render_arrays() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    device_render_cells[i] = 0;
  }
}


#ifdef OPTION_RENDER
__device__ void Alive::update_render_array() {
  device_render_cells[cell_id_] = state_ + 1;
}

__device__ void Candidate::update_render_array() {
  device_render_cells[cell_id_] = -1;
}

void render() {
  initialize_render_arrays<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
  allocator_handle->parallel_do<Alive, &Alive::update_render_array>();
  allocator_handle->parallel_do<Candidate, &Candidate::update_render_array>();
  
  hipMemcpy(host_render_cells, d_device_render_cells,
             sizeof(int)*dataset.x*dataset.y, hipMemcpyDeviceToHost);
  draw(host_render_cells);
}
#endif  // OPTION_RENDER

void transfer_dataset() {
  int* dev_cell_ids;
  int num_alive = dataset.alive_cells.size();
  hipMalloc(&dev_cell_ids, sizeof(int)*num_alive);
  hipMemcpy(dev_cell_ids, dataset.alive_cells.data(), sizeof(int)*num_alive,
             hipMemcpyHostToDevice);

#ifndef NDEBUG
  printf("Loading on GPU: %i alive cells.\n", num_alive);
#endif  // NDEBUG

  load_game<<<1024, 1024>>>(dev_cell_ids, num_alive);
  gpuErrchk(hipDeviceSynchronize());
  hipFree(dev_cell_ids);

  allocator_handle->parallel_do<Alive, &Alive::update>();
}


__device__ int device_checksum;
__device__ int device_num_candidates;

__device__ void Alive::update_checksum() {
  atomicAdd(&device_checksum, 1);
}


__device__ void Candidate::update_counter() {
  atomicAdd(&device_num_candidates, 1);
}

int checksum() {
  int host_checksum = 0;
  int host_num_candidates = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(device_checksum), &host_checksum, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(device_num_candidates), &host_num_candidates, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  allocator_handle->parallel_do<Alive, &Alive::update_checksum>();
  allocator_handle->parallel_do<Candidate, &Candidate::update_counter>();

  hipMemcpyFromSymbol(&host_checksum, HIP_SYMBOL(device_checksum), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(&host_num_candidates, HIP_SYMBOL(device_num_candidates), sizeof(int), 0,
                       hipMemcpyDeviceToHost);

  return host_checksum;
}


#ifdef OPTION_DEFRAG
void defrag() {
  allocator_handle->parallel_defrag<Alive>();
  allocator_handle->parallel_defrag<Candidate>();
}
#endif  // OPTION_DEFRAG


int main(int /*argc*/, char** /*argv*/) {
  // Load data set.
  dataset = load_burst();

  hipMemcpyToSymbol(HIP_SYMBOL(SIZE_X), &dataset.x, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(SIZE_Y), &dataset.y, sizeof(int), 0,
                     hipMemcpyHostToDevice);

#ifdef OPTION_RENDER
  init_renderer();
#endif  // OPTION_RENDER

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Allocate memory.
  Cell** host_cells;
  hipMalloc(&host_cells, sizeof(Cell*)*dataset.x*dataset.y);
  hipMemcpyToSymbol(HIP_SYMBOL(cells), &host_cells, sizeof(Cell**), 0,
                     hipMemcpyHostToDevice);

#ifdef OPTION_RENDER
  hipMalloc(&d_device_render_cells, sizeof(int)*dataset.x*dataset.y);
  hipMemcpyToSymbol(HIP_SYMBOL(device_render_cells), &d_device_render_cells,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  host_render_cells = new int[dataset.x*dataset.y];
#endif  // OPTION_RENDER

  // Initialize cells.
  create_cells<<<1024, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());

  transfer_dataset();

  auto time_start = std::chrono::system_clock::now();

  // Run simulation.
  for (int i = 0; i < kNumIterations; ++i) {
    if (i % 50 == 0) {
      defrag();
    }

#ifdef OPTION_RENDER
    render();
#endif  // OPTION_RENDER

#ifndef NDEBUG
    printf("%i\n", i);
    allocator_handle->DBG_print_state_stats();
    // allocator_handle->DBG_collect_stats();
#endif  // NDEBUG

    allocator_handle->parallel_do<Candidate, &Candidate::prepare>();
    allocator_handle->parallel_do<Alive, &Alive::prepare>();
    allocator_handle->parallel_do<Candidate, &Candidate::update>();
    allocator_handle->parallel_do<Alive, &Alive::update>();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto micros = std::chrono::duration_cast<std::chrono::microseconds>(elapsed)
      .count();

#ifdef OPTION_RENDER
  close_renderer();
#endif  // OPTION_RENDER

#ifndef NDEBUG
  printf("Checksum: %i\n", checksum());
#endif  // NDEBUG

  printf("%lu, %lu\n", micros, allocator_handle->DBG_get_enumeration_time());

//  if (kOptionPrintStats) {
//    allocator_handle->DBG_print_collected_stats();
//  }

#ifdef OPTION_RENDER
  delete[] host_render_cells;
  hipFree(d_device_render_cells);
#endif  // OPTION_RENDER

  hipFree(host_cells);

  return 0;
}
