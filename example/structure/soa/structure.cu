#include "hip/hip_runtime.h"
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "../dataset.h"
#include "../rendering.h"
#include "structure.h"


// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;


__device__ NodeBase::NodeBase(float pos_x, float pos_y)
    : pos_x_(pos_x), pos_y_(pos_y), num_springs_(0) {}


__device__ AnchorNode::AnchorNode(float pos_x, float pos_y)
    : NodeBase(pos_x, pos_y) {}


__device__ AnchorPullNode::AnchorPullNode(float pos_x, float pos_y,
                                          float vel_x, float vel_y)
    : AnchorNode(pos_x, pos_y), vel_x_(vel_x), vel_y_(vel_y) {}


__device__ Node::Node(float pos_x, float pos_y, float mass)
    : NodeBase(pos_x, pos_y), mass_(mass), vel_x_(0.0f), vel_y_(0.0f) {}


__device__ Spring::Spring(NodeBase* p1, NodeBase* p2, float spring_factor,
                          float max_force)
    : p1_(p1), p2_(p2), spring_factor_(spring_factor), force_(0.0f),
      max_force_(max_force), initial_length_(p1->distance_to(p2)),
      delete_flag_(0) {
  assert(initial_length_ > 0.0f);
  p1_->add_spring(this);
  p2_->add_spring(this);
}


// Only used during graph creation.
__device__ void NodeBase::add_spring(Spring* spring) {
  int idx = atomicAdd(&num_springs_, 1);
  springs_[idx] = spring;
  assert(idx + 1 <= kMaxDegree);
  assert(spring->p1() == this || spring->p2() == this);
}


__device__ void NodeBase::remove_spring(Spring* spring) {
  for (int i = 0; i < kMaxDegree; ++i) {
    if (springs_[i] == spring) {
      springs_[i] = nullptr;
      if (atomicSub(&num_springs_, 1) == 1) {
        // Deleted last spring.
        destroy(device_allocator, this);
      }
      return;
    }
  }

  // Spring not found.
  assert(false);
}


__device__ float NodeBase::distance_to(NodeBase* other) const {
  float dx = pos_x_ - other->pos_x_;
  float dy = pos_y_ - other->pos_y_;
  float dist_sq = dx*dx + dy*dy;
  return sqrt(dist_sq);
}


__device__ void AnchorPullNode::pull() {
  pos_x_ += vel_x_ * kDt;
  pos_y_ += vel_y_ * kDt;
}


__device__ void Spring::compute_force() {
  float dist = p1_->distance_to(p2_);
  float displacement = max(0.0f, dist - initial_length_);
  force_ = spring_factor_ * displacement;

  if (force_ > max_force_) { self_destruct(); }
}


__device__ void Spring::self_destruct() {
  p1_->remove_spring(this);
  p2_->remove_spring(this);
  destroy(device_allocator, this);
}


__device__ void Node::compute_force() {
  force_x_ = 0.0f;
  force_y_ = 0.0f;

  for (int i = 0; i < kMaxDegree; ++i) {
    Spring* s = springs_[i];
    if (s != nullptr) {
      NodeBase* from;
      NodeBase* to;

      if (s->p1() == this) {
        from = this;
        to = s->p2();
      } else {
        assert(s->p2() == this);
        from = this;
        to = s->p1();
      }

      // Calculate unit vector.
      float dx = to->pos_x() - from->pos_x();
      float dy = to->pos_y() - from->pos_y();
      float dist = sqrt(dx*dx + dy*dy);
      float unit_x = dx/dist;
      float unit_y = dy/dist;

      // Apply force.
      force_x_ += unit_x*s->force();
      force_y_ += unit_y*s->force();
    }
  }
}


__device__ void Node::move() {
  // Calculate new velocity and posFition.
  vel_x_ += force_x_*kDt / mass_;
  vel_y_ += force_y_*kDt / mass_;
  vel_x_ *= 1.0f - kVelocityDampening;
  vel_y_ *= 1.0f - kVelocityDampening;
  pos_x_ += vel_x_*kDt;
  pos_y_ += vel_y_*kDt;
}


__device__ void NodeBase::initialize_bfs() {
  if (this->cast<AnchorNode>() != nullptr) {
    distance_ = 0;
  } else {
    distance_ = kMaxDistance;  // should be int_max
  }
}


__device__ bool dev_bfs_continue;

__device__ void NodeBase::bfs_visit(int distance) {
  if (distance == distance_) {
    // Continue until all vertices were visited.
    dev_bfs_continue = true;

    for (int i = 0; i < kMaxDegree; ++i) {
      auto* spring = springs_[i];

      if (spring != nullptr) {
        // Find neighboring vertices.
        NodeBase* n;
        if (this == spring->p1()) {
          n = spring->p2();
        } else {
          n = spring->p1();
        }

        if (n->distance_ == kMaxDistance) {
          // Set distance on neighboring vertex if unvisited.
          n->distance_ = distance + 1;
        }
      }
    }
  }
}


__device__ void NodeBase::bfs_set_delete_flags() {
  if (distance_ == kMaxDistance) {  // should be int_max
    for (int i = 0; i < kMaxDegree; ++i) {
      auto* spring = springs_[i];
      if (spring != nullptr) {
        spring->set_delete_flag();
      }
    }
  }
}


__device__ void Spring::bfs_delete() {
  if (delete_flag_ == 1) { self_destruct(); }
}


// Only for rendering.
__device__ int dev_num_springs;
__device__ SpringInfo dev_spring_info[kMaxSprings];
int host_num_springs;
SpringInfo host_spring_info[kMaxSprings];

__device__ void Spring::add_to_rendering_array() {
  int idx = atomicAdd(&dev_num_springs, 1);
  dev_spring_info[idx].p1_x = p1_->pos_x();
  dev_spring_info[idx].p1_y = p1_->pos_y();
  dev_spring_info[idx].p2_x = p2_->pos_x();
  dev_spring_info[idx].p2_y = p2_->pos_y();
  dev_spring_info[idx].force = force_;
  dev_spring_info[idx].max_force = max_force_;
}


void transfer_data() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_springs), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  allocator_handle->parallel_do<Spring, &Spring::add_to_rendering_array>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(&host_num_springs, HIP_SYMBOL(dev_num_springs), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(host_spring_info, HIP_SYMBOL(dev_spring_info),
                       sizeof(SpringInfo)*host_num_springs, 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


float checksum() {
  transfer_data();
  float result = 0.0f;

  for (int i = 0; i < host_num_springs; ++i) {
    result += host_spring_info[i].p1_x*host_spring_info[i].p2_y
              *host_spring_info[i].force;
  }

  return result;
}


void compute() {
  allocator_handle->parallel_do<Spring, &Spring::compute_force>();
  allocator_handle->parallel_do<Node, &Node::compute_force>();
  allocator_handle->parallel_do<Node, &Node::move>();
}


void bfs_and_delete() {
  // Perform BFS to check reachability.
  allocator_handle->parallel_do<NodeBase, &NodeBase::initialize_bfs>();

  for (int i = 0; i < kMaxDistance; ++i) {
    bool continue_flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(dev_bfs_continue), &continue_flag, sizeof(bool), 0,
                       hipMemcpyHostToDevice);
    allocator_handle->parallel_do<NodeBase, int, &NodeBase::bfs_visit>(i);
    hipMemcpyFromSymbol(&continue_flag, HIP_SYMBOL(dev_bfs_continue), sizeof(bool), 0,
                         hipMemcpyDeviceToHost);

    if (!continue_flag) break;
  }

  // Delete springs (and nodes).
  allocator_handle->parallel_do<NodeBase, &NodeBase::bfs_set_delete_flags>();
  allocator_handle->parallel_do<Spring, &Spring::bfs_delete>();
}


#ifdef OPTION_DEFRAG
void defrag() {
  allocator_handle->parallel_defrag<AnchorNode>(1);
  allocator_handle->parallel_defrag<AnchorPullNode>(1);
  allocator_handle->parallel_defrag<Node>(1);
  allocator_handle->parallel_defrag<Spring>(1);
}
#endif  // OPTION_DEFRAG


void step() {
  allocator_handle->parallel_do<AnchorPullNode, &AnchorPullNode::pull>();

  for (int i = 0; i < kNumComputeIterations; ++i) {
    compute();
  }

  bfs_and_delete();

  if (kOptionRender) {
    transfer_data();
    draw(host_num_springs, host_spring_info);
  }
}


__device__ NodeBase* tmp_nodes[kMaxNodes];

__global__ void kernel_create_nodes(DsNode* nodes, int num_nodes) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_nodes; i += blockDim.x * gridDim.x) {
    if (nodes[i].type == kTypeNode) {
      tmp_nodes[i] = new(device_allocator) Node(nodes[i].pos_x,
                                                nodes[i].pos_y,
                                                nodes[i].mass);
    } else if (nodes[i].type == kTypeAnchorPullNode) {
      tmp_nodes[i] = new(device_allocator) AnchorPullNode(nodes[i].pos_x,
                                                          nodes[i].pos_y,
                                                          nodes[i].vel_x,
                                                          nodes[i].vel_y);
    } else if (nodes[i].type == kTypeAnchorNode) {
      tmp_nodes[i] = new(device_allocator) AnchorNode(nodes[i].pos_x,
                                                      nodes[i].pos_y);
    } else {
      assert(false);
    }
  }
}


__global__ void kernel_create_springs(DsSpring* springs, int num_springs) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_springs; i += blockDim.x * gridDim.x) {
    assert(tmp_nodes[springs[i].p1] != nullptr);
    assert(tmp_nodes[springs[i].p2] != nullptr);

    new(device_allocator) Spring(tmp_nodes[springs[i].p1],
                                 tmp_nodes[springs[i].p2],
                                 springs[i].spring_factor,
                                 springs[i].max_force);
  }
}


void load_dataset(Dataset& dataset) {
  DsNode* host_nodes;
  hipMalloc(&host_nodes, sizeof(DsNode)*dataset.nodes.size());
  hipMemcpy(host_nodes, dataset.nodes.data(),
             sizeof(DsNode)*dataset.nodes.size(), hipMemcpyHostToDevice);

  DsSpring* host_springs;
  hipMalloc(&host_springs, sizeof(DsSpring)*dataset.springs.size());
  hipMemcpy(host_springs, dataset.springs.data(),
             sizeof(DsSpring)*dataset.springs.size(), hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_nodes<<<128, 128>>>(host_nodes, dataset.nodes.size());
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_springs<<<128, 128>>>(host_springs, dataset.springs.size());
  gpuErrchk(hipDeviceSynchronize());

  hipFree(host_nodes);
  hipFree(host_springs);
}


__global__ void load_example() {
  assert(threadIdx.x == 0 && blockIdx.x == 0);

  float spring_factor = 5.0f;
  float max_force = 100.0f;
  float mass = 500.0f;

  auto* a1 = new(device_allocator) AnchorPullNode(0.1, 0.5, 0.0, -0.02);
  auto* a2 = new(device_allocator) AnchorPullNode(0.3, 0.5, 0.0, -0.02);
  auto* a3 = new(device_allocator) AnchorPullNode(0.5, 0.5, 0.0, -0.02);

  auto* n1 = new(device_allocator) Node(0.05, 0.6, mass);
  auto* n2 = new(device_allocator) Node(0.3, 0.6, mass);
  auto* n3 = new(device_allocator) Node(0.7, 0.6, mass);

  auto* n4 = new(device_allocator) Node(0.2, 0.7, mass);
  auto* n5 = new(device_allocator) Node(0.4, 0.7, mass);
  auto* n6 = new(device_allocator) Node(0.8, 0.7, mass);

  auto* a4 = new(device_allocator) AnchorNode(0.1, 0.9);
  auto* a5 = new(device_allocator) AnchorNode(0.3, 0.9);
  auto* a6 = new(device_allocator) AnchorNode(0.6, 0.9);

  new(device_allocator) Spring(a1, n1, spring_factor, max_force);
  new(device_allocator) Spring(a2, n2, spring_factor, max_force);
  new(device_allocator) Spring(a3, n3, spring_factor, max_force);

  new(device_allocator) Spring(n1, n4, spring_factor, max_force);
  new(device_allocator) Spring(n2, n5, spring_factor, max_force);
  new(device_allocator) Spring(n3, n6, spring_factor, max_force);
  new(device_allocator) Spring(n2, n6, spring_factor, max_force);

  new(device_allocator) Spring(n4, a4, spring_factor, max_force);
  new(device_allocator) Spring(n5, a5, spring_factor, max_force);
  new(device_allocator) Spring(n6, a6, spring_factor, max_force);
}


int main(int /*argc*/, char** /*argv*/) {
  if (kOptionRender) {
    init_renderer();
  }

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  //load_example<<<1, 1>>>();
  
  Dataset dataset;
  random_dataset(dataset);
  load_dataset(dataset);

//  allocator_handle->DBG_print_state_stats();

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumSteps; ++i) {
   // printf("%i\n", i);

#ifdef OPTION_DEFRAG
    if (kOptionDefrag && i % 20 == 0) {
      // allocator_handle->DBG_print_state_stats();
      defrag();
    }
#endif  // OPTION_DEFRAG

/*
    int a_anchornode = dev_ptr->DBG_host_allocated_slots<AnchorNode>();
    int u_anchornode = dev_ptr->DBG_host_used_slots<AnchorNode>();
    int a_anchorpullnode = dev_ptr->DBG_host_allocated_slots<AnchorPullNode>();
    int u_anchorpullnode = dev_ptr->DBG_host_used_slots<AnchorPullNode>();
    int a_node = dev_ptr->DBG_host_allocated_slots<Node>();
    int u_node = dev_ptr->DBG_host_used_slots<Node>();
    int a_spring = dev_ptr->DBG_host_allocated_slots<Spring>();
    int u_spring = dev_ptr->DBG_host_used_slots<Spring>();
    printf("%i, %i, %i, %i, %i, %i, %i, %i, %i\n",
           i, a_anchornode, u_anchornode, a_anchorpullnode, u_anchorpullnode,
           a_node, u_node, a_spring, u_spring);
*/

    step();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("%lu,%lu\n", millis, allocator_handle->DBG_get_enumeration_time());

  allocator_handle->DBG_print_defrag_time();

  if (kOptionPrintStats) {
    //allocator_handle->DBG_print_state_stats();
  }

#ifndef NDEBUG
  printf("Checksum: %f\n", checksum());
#endif  // NDEBUG

  if (kOptionRender) {
    close_renderer();
  }
}
