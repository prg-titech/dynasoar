#include "hip/hip_runtime.h"
#include <assert.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <limits>
#include <stdio.h>

#include "../configuration.h"
#include "../dataset.h"
#include "util/util.h"


static const int kThreads = 256;
static const int kNullptr = std::numeric_limits<int>::max();

using IndexT = int;

struct Node {
  DeviceArray<IndexT, kMaxDegree> springs;
  int num_springs;
  int distance;
  float pos_x;
  float pos_y;
  float vel_x;
  float vel_y;
  float mass;
  char type;
};

struct Spring {
  IndexT p1;
  IndexT p2;
  float factor;
  float initial_length;
  float force;
  float max_force;
  bool is_active;
  bool delete_flag;
};

__device__ Node* dev_nodes;
__device__ Spring* dev_springs;


__device__ void new_NodeBase(IndexT id, float pos_x, float pos_y) {
  dev_nodes[id].pos_x = pos_x;
  dev_nodes[id].pos_y = pos_y;
  dev_nodes[id].num_springs = 0;
  dev_nodes[id].type = kTypeNodeBase;

  for (int i = 0; i < kMaxDegree; ++i) {
    dev_nodes[id].springs[i] = kNullptr;
  }
}


__device__ void new_AnchorNode(IndexT id, float pos_x, float pos_y) {
  new_NodeBase(id, pos_x, pos_y);
  dev_nodes[id].type = kTypeAnchorNode;
}


__device__ void new_AnchorPullNode(IndexT id, float pos_x, float pos_y,
                                   float vel_x, float vel_y) {
  new_AnchorNode(id, pos_x, pos_y);
  dev_nodes[id].vel_x = vel_x;
  dev_nodes[id].vel_y = vel_y;
  dev_nodes[id].type = kTypeAnchorPullNode;
}


__device__ void new_Node(IndexT id, float pos_x, float pos_y, float mass) {
  new_NodeBase(id, pos_x, pos_y);
  dev_nodes[id].mass = mass;
  dev_nodes[id].type = kTypeNode;
}


__device__ float NodeBase_distance_to(IndexT id, IndexT other) {
  float dx = dev_nodes[id].pos_x - dev_nodes[other].pos_x;
  float dy = dev_nodes[id].pos_y - dev_nodes[other].pos_y;
  float dist_sq = dx*dx + dy*dy;
  return sqrt(dist_sq);
}


__device__ void NodeBase_add_spring(IndexT id, IndexT spring) {
  assert(id >= 0 && id < kMaxNodes);

  int idx = atomicAdd(&dev_nodes[id].num_springs, 1);
  assert(idx + 1 <= kMaxDegree);
  dev_nodes[id].springs[idx] = spring;

  assert(dev_springs[spring].p1 == id || dev_springs[spring].p2 == id);
}


__device__ void new_Spring(IndexT id, IndexT p1, IndexT p2,
                           float spring_factor, float max_force) {
  dev_springs[id].is_active = true;
  dev_springs[id].p1 = p1;
  dev_springs[id].p2 = p2;
  dev_springs[id].factor = spring_factor;
  dev_springs[id].force = 0.0f;
  dev_springs[id].max_force = max_force;
  dev_springs[id].initial_length = NodeBase_distance_to(p1, p2);
  dev_springs[id].delete_flag = false;
  assert(dev_springs[id].initial_length > 0.0f);

  NodeBase_add_spring(p1, id);
  NodeBase_add_spring(p2, id);
}


__device__ void NodeBase_remove_spring(IndexT id, IndexT spring) {
  for (int i = 0; i < kMaxDegree; ++i) {
    if (dev_nodes[id].springs[i] == spring) {
      dev_nodes[id].springs[i] = kNullptr;
      if (atomicSub(&dev_nodes[id].num_springs, 1) == 1) {
        // Deleted last spring.
        dev_nodes[id].type = 0;
      }
      return;
    }
  }

  // Spring not found.
  assert(false);
}


__device__ void AnchorPullNode_pull(IndexT id) {
  dev_nodes[id].pos_x += dev_nodes[id].vel_x * kDt;
  dev_nodes[id].pos_y += dev_nodes[id].vel_y * kDt;
}


__device__ void Spring_self_destruct(IndexT id) {
  NodeBase_remove_spring(dev_springs[id].p1, id);
  NodeBase_remove_spring(dev_springs[id].p2, id);
  dev_springs[id].is_active = false;
}


__device__ void Spring_compute_force(IndexT id) {
  float dist = NodeBase_distance_to(dev_springs[id].p1, dev_springs[id].p2);
  float displacement = max(0.0f, dist - dev_springs[id].initial_length);
  dev_springs[id].force = dev_springs[id].factor * displacement;

  if (dev_springs[id].force > dev_springs[id].max_force) {
    Spring_self_destruct(id);
  }
}


__device__ void Node_move(IndexT id) {
  float force_x = 0.0f;
  float force_y = 0.0f;

  for (int i = 0; i < kMaxDegree; ++i) {
    IndexT s = dev_nodes[id].springs[i];

    if (s != kNullptr) {
      IndexT from;
      IndexT to;

      if (dev_springs[s].p1 == id) {
        from = id;
        to = dev_springs[s].p2;
      } else {
        assert(dev_springs[s].p2 == id);
        from = id;
        to = dev_springs[s].p1;
      }

      // Calculate unit vector.
      float dx = dev_nodes[to].pos_x - dev_nodes[from].pos_x;
      float dy = dev_nodes[to].pos_y - dev_nodes[from].pos_y;
      float dist = sqrt(dx*dx + dy*dy);
      float unit_x = dx/dist;
      float unit_y = dy/dist;

      // Apply force.
      force_x += unit_x*dev_springs[s].force;
      force_y += unit_y*dev_springs[s].force;
    }
  }

  // Calculate new velocity and position.
  dev_nodes[id].vel_x += force_x*kDt / dev_nodes[id].mass;
  dev_nodes[id].vel_y += force_y*kDt / dev_nodes[id].mass;
  dev_nodes[id].vel_x *= 1.0f - kVelocityDampening;
  dev_nodes[id].vel_y *= 1.0f - kVelocityDampening;
  dev_nodes[id].pos_x += dev_nodes[id].vel_x*kDt;
  dev_nodes[id].pos_y += dev_nodes[id].vel_y*kDt;
}


__device__ void NodeBase_initialize_bfs(IndexT id) {
  if (dev_nodes[id].type == kTypeAnchorNode) {
    dev_nodes[id].distance = 0;
  } else {
    dev_nodes[id].distance = kMaxDistance;  // should be int_max
  }
}


__device__ bool dev_bfs_continue;

__device__ void NodeBase_bfs_visit(IndexT id, int distance) {
  if (distance == dev_nodes[id].distance) {
    // Continue until all vertices were visited.
    dev_bfs_continue = true;

    for (int i = 0; i < kMaxDegree; ++i) {
      IndexT spring = dev_nodes[id].springs[i];

      if (spring != kNullptr) {
        // Find neighboring vertices.
        IndexT n;
        if (id == dev_springs[spring].p1) {
          n = dev_springs[spring].p2;
        } else {
          n = dev_springs[spring].p1;
        }

        if (dev_nodes[n].distance == kMaxDistance) {
          // Set distance on neighboring vertex if unvisited.
          dev_nodes[n].distance = distance + 1;
        }
      }
    }
  }
}


__device__ void NodeBase_bfs_set_delete_flags(IndexT id) {
  if (dev_nodes[id].distance == kMaxDistance) {  // should be int_max
    for (int i = 0; i < kMaxDegree; ++i) {
      IndexT spring = dev_nodes[id].springs[i];
      if (spring != kNullptr) {
        dev_springs[spring].delete_flag = true;
      }
    }
  }
}


__device__ void Spring_bfs_delete(IndexT id) {
  if (dev_springs[id].delete_flag) { Spring_self_destruct(id); }
}


// Only for rendering and checksum computation.
__device__ int dev_num_springs;
__device__ SpringInfo dev_spring_info[kMaxSprings];
int host_num_springs;
SpringInfo host_spring_info[kMaxSprings];

__device__ void Spring_add_to_rendering_array(IndexT id) {
  int idx = atomicAdd(&dev_num_springs, 1);
  dev_spring_info[idx].p1_x = dev_nodes[dev_springs[id].p1].pos_x;
  dev_spring_info[idx].p1_y = dev_nodes[dev_springs[id].p1].pos_y;
  dev_spring_info[idx].p2_x = dev_nodes[dev_springs[id].p2].pos_x;
  dev_spring_info[idx].p2_y = dev_nodes[dev_springs[id].p2].pos_y;
  dev_spring_info[idx].force = dev_springs[id].force;
  dev_spring_info[idx].max_force = dev_springs[id].max_force;
}


__global__ void kernel_AnchorPullNode_pull() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_nodes[i].type == kTypeAnchorPullNode) {
      AnchorPullNode_pull(i);
    }
  }
}


__global__ void kernel_Node_move() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_nodes[i].type == kTypeNode) {
      Node_move(i);
    }
  }
}


__global__ void kernel_NodeBase_initialize_bfs() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_nodes[i].type != 0) {
      NodeBase_initialize_bfs(i);
    }
  }
}


__global__ void kernel_NodeBase_bfs_visit(int dist) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_nodes[i].type != 0) {
      NodeBase_bfs_visit(i, dist);
    }
  }
}


__global__ void kernel_NodeBase_bfs_set_delete_flags() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_nodes[i].type != 0) {
      NodeBase_bfs_set_delete_flags(i);
    }
  }
}


__global__ void kernel_Spring_compute_force() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    if (dev_springs[i].is_active) {
      Spring_compute_force(i);
    }
  }
}


__global__ void kernel_Spring_bfs_delete() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    if (dev_springs[i].is_active) {
      Spring_bfs_delete(i);
    }
  }
}


__global__ void kernel_Spring_add_to_rendering_array() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    if (dev_springs[i].is_active) {
      Spring_add_to_rendering_array(i);
    }
  }
}


__global__ void kernel_initialize_nodes() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    dev_nodes[i].type = 0;
  }
}


__global__ void kernel_initialize_springs() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    dev_springs[i].is_active = false;
  }
}


void transfer_data() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_springs), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_Spring_add_to_rendering_array<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(&host_num_springs, HIP_SYMBOL(dev_num_springs), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(host_spring_info, HIP_SYMBOL(dev_spring_info),
                       sizeof(SpringInfo)*host_num_springs, 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


float checksum() {
  transfer_data();
  float result = 0.0f;

  for (int i = 0; i < host_num_springs; ++i) {
    result += host_spring_info[i].p1_x*host_spring_info[i].p2_y
              *host_spring_info[i].force;
  }

  return result;
}


void compute() {
  kernel_Spring_compute_force<<<(kMaxSprings + kThreads - 1) / kThreads,
                                kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Node_move<<<(kMaxNodes + kThreads - 1) / kThreads,
                     kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void bfs_and_delete() {
  // Perform BFS to check reachability.
  kernel_NodeBase_initialize_bfs<<<(kMaxNodes + kThreads - 1) / kThreads,
                                    kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < kMaxDistance; ++i) {
    bool continue_flag = false;
    hipMemcpyToSymbol(HIP_SYMBOL(dev_bfs_continue), &continue_flag, sizeof(bool), 0,
                       hipMemcpyHostToDevice);

    kernel_NodeBase_bfs_visit<<<(kMaxNodes + kThreads - 1) / kThreads,
                                kThreads>>>(i);
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpyFromSymbol(&continue_flag, HIP_SYMBOL(dev_bfs_continue), sizeof(bool), 0,
                         hipMemcpyDeviceToHost);

    if (!continue_flag) break;
  }

  // Delete springs (and nodes).
  kernel_NodeBase_bfs_set_delete_flags<<<(kMaxNodes + kThreads - 1) / kThreads,
                                         kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Spring_bfs_delete<<<(kMaxSprings + kThreads - 1) / kThreads,
                             kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void step() {
  kernel_AnchorPullNode_pull<<<(kMaxNodes + kThreads - 1) / kThreads,
                               kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < kNumComputeIterations; ++i) {
    compute();
  }

  bfs_and_delete();
}


void initialize_memory() {
  kernel_initialize_nodes<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_initialize_springs<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


__device__ IndexT dev_tmp_nodes[kMaxNodes];
__device__ IndexT dev_node_counter;
__global__ void kernel_create_nodes(DsNode* nodes, int num_nodes) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_nodes; i += blockDim.x * gridDim.x) {
    int idx = atomicAdd(&dev_node_counter, 1);
    dev_tmp_nodes[i] = idx;

    if (nodes[i].type == kTypeNode) {
      new_Node(idx, nodes[i].pos_x, nodes[i].pos_y, nodes[i].mass);
    } else if (nodes[i].type == kTypeAnchorPullNode) {
      new_AnchorPullNode(idx, nodes[i].pos_x, nodes[i].pos_y, nodes[i].vel_x,
                         nodes[i].vel_y);
    } else if (nodes[i].type == kTypeAnchorNode) {
      new_AnchorNode(idx, nodes[i].pos_x, nodes[i].pos_y);
    } else {
      assert(false);
    }
  }
}


__global__ void kernel_create_springs(DsSpring* springs, int num_springs) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_springs; i += blockDim.x * gridDim.x) {
    new_Spring(i, dev_tmp_nodes[springs[i].p1], dev_tmp_nodes[springs[i].p2],
               springs[i].spring_factor, springs[i].max_force);
  }
}


void load_dataset(Dataset& dataset) {
  DsNode* host_nodes;
  hipMalloc(&host_nodes, sizeof(DsNode)*dataset.nodes.size());
  hipMemcpy(host_nodes, dataset.nodes.data(),
             sizeof(DsNode)*dataset.nodes.size(), hipMemcpyHostToDevice);

  DsSpring* host_springs;
  hipMalloc(&host_springs, sizeof(DsSpring)*dataset.springs.size());
  hipMemcpy(host_springs, dataset.springs.data(),
             sizeof(DsSpring)*dataset.springs.size(), hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  IndexT zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_node_counter), &zero, sizeof(IndexT), 0,
                     hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_nodes<<<128, 128>>>(host_nodes, dataset.nodes.size());
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_springs<<<128, 128>>>(host_springs, dataset.springs.size());
  gpuErrchk(hipDeviceSynchronize());

  hipFree(host_nodes);
  hipFree(host_springs);
}


int main(int /*argc*/, char** /*argv*/) {
  // Allocate memory.
  Node* host_nodes;
  hipMalloc(&host_nodes, sizeof(Node)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_nodes), &host_nodes, sizeof(Node*), 0,
                     hipMemcpyHostToDevice);

  Spring* host_springs;
  hipMalloc(&host_springs, sizeof(Spring)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_springs), &host_springs, sizeof(Spring*), 0,
                     hipMemcpyHostToDevice);

  initialize_memory();

  Dataset dataset;
  random_dataset(dataset);
  load_dataset(dataset);

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumSteps; ++i) {
#ifndef NDEBUG
    printf("%i\n", i);
#endif  // NDEBUG
    step();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto micros = std::chrono::duration_cast<std::chrono::microseconds>(elapsed)
      .count();

  printf("%lu\n", micros);

#ifndef NDEBUG
  printf("Checksum: %f\n", checksum());
#endif  // NDEBUG
}
