#include "hip/hip_runtime.h"
#include <assert.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <limits>
#include <stdio.h>

#include "../configuration.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static const int kThreads = 256;
static const int kBlocks = (kSize*kSize + kThreads - 1)/kThreads;
static const int kNullptr = std::numeric_limits<int>::max();

static const char kNoType = 0;
static const char kClassMale = 1;
static const char kClassFemale = 2;

__device__ hiprandState_t* dev_Cell_random_state;
// (No field for agent)
__device__ int* dev_Cell_sugar_diffusion;
__device__ int* dev_Cell_sugar;
__device__ int* dev_Cell_sugar_capacity;
__device__ int* dev_Cell_grow_rate;
// (No field for cell_id)
__device__ char* dev_Cell_Agent_type;
__device__ hiprandState_t* dev_Cell_Agent_random_state;
// (No field for cell)
__device__ int* dev_Cell_Agent_cell_request;
__device__ int* dev_Cell_Agent_vision;
__device__ int* dev_Cell_Agent_age;
__device__ int* dev_Cell_Agent_max_age;
__device__ int* dev_Cell_Agent_sugar;
__device__ int* dev_Cell_Agent_metabolism;
__device__ int* dev_Cell_Agent_endowment;
__device__ bool* dev_Cell_Agent_permission;
__device__ int* dev_Cell_Male_female_request;
__device__ bool* dev_Cell_Male_proposal_accepted;
__device__ int* dev_Cell_Female_num_children;
__device__ int* dev_Cell_Female_max_children;


__device__ float Cell_random_float(int cell_id) {
  return hiprand_uniform(&dev_Cell_random_state[cell_id]);
}


__device__ int Cell_random_int(int cell_id, int a, int b) {
  return hiprand(&dev_Cell_random_state[cell_id]) % (b - a) + a;
}


__device__ float Agent_random_float(int cell_id) {
  return hiprand_uniform(&dev_Cell_Agent_random_state[cell_id]);
}


__device__ bool Cell_is_free(int cell_id) {
  return dev_Cell_Agent_type[cell_id] == kNoType;
}


__device__ void Cell_enter(int cell_id, int agent) {
  assert(dev_Cell_Agent_type[cell_id] == kNoType);
  assert(dev_Cell_Agent_type[agent] != kNoType);

  dev_Cell_Agent_permission[cell_id] = false;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;

  // Threadfence to make sure that cell will not be processed by accident.
  // E.g.: permission set to false first before setting new type.
  __threadfence();

  dev_Cell_Agent_type[cell_id] = dev_Cell_Agent_type[agent];

  __threadfence();

  dev_Cell_Agent_random_state[cell_id] = dev_Cell_Agent_random_state[agent];
  dev_Cell_Agent_vision[cell_id] = dev_Cell_Agent_vision[agent];
  dev_Cell_Agent_age[cell_id] = dev_Cell_Agent_age[agent];
  dev_Cell_Agent_max_age[cell_id] = dev_Cell_Agent_max_age[agent];
  dev_Cell_Agent_sugar[cell_id] = dev_Cell_Agent_sugar[agent];
  dev_Cell_Agent_metabolism[cell_id] = dev_Cell_Agent_metabolism[agent];
  dev_Cell_Agent_endowment[cell_id] = dev_Cell_Agent_endowment[agent];
  dev_Cell_Female_max_children[cell_id] = dev_Cell_Female_max_children[agent];
  dev_Cell_Female_num_children[cell_id] = dev_Cell_Female_num_children[agent];
}


__device__ void Cell_leave(int cell_id) {
  assert(dev_Cell_Agent_type[cell_id] != kNoType);
  dev_Cell_Agent_type[cell_id] = kNoType;
  dev_Cell_Agent_permission[cell_id] = false;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
}


__device__ void Agent_harvest_sugar(int cell_id) {
  // Harvest as much sugar as possible.
  // TODO: Do we need two sugar fields here?
  dev_Cell_Agent_sugar[cell_id] += dev_Cell_sugar[cell_id];
  dev_Cell_sugar[cell_id] = 0;
}


__device__ bool Agent_ready_to_mate(int cell_id) {
  // Half of endowment of sugar will go to the child. And the parent still
  // needs some sugar to survive.
  return (dev_Cell_Agent_sugar[cell_id]
          >= dev_Cell_Agent_endowment[cell_id] * 2 / 3)
      && dev_Cell_Agent_age[cell_id] >= kMinMatingAge;
}


__device__ void new_Cell(int cell_id, int seed, int sugar, int sugar_capacity,
                         int max_grow_rate) {
  dev_Cell_sugar[cell_id] = sugar;
  dev_Cell_sugar_capacity[cell_id] = sugar_capacity;

  hiprand_init(seed, cell_id, 0, &dev_Cell_random_state[cell_id]);

  // Set random grow rate.
  float r = hiprand_uniform(&dev_Cell_random_state[cell_id]);

  if (r <= 0.02) {
    dev_Cell_grow_rate[cell_id] = max_grow_rate;
  } else if (r <= 0.04) {
    dev_Cell_grow_rate[cell_id] = 0.5*max_grow_rate;
  } else if (r <= 0.08) {
    dev_Cell_grow_rate[cell_id] = 0.25*max_grow_rate;
  } else {
    dev_Cell_grow_rate[cell_id] = 0;
  }
}


__device__ void new_Agent(int cell_id, int vision, int age, int max_age,
                          int endowment, int metabolism) {
  assert(cell_id != kNullptr);
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
  dev_Cell_Agent_vision[cell_id] = vision;
  dev_Cell_Agent_age[cell_id] = age;
  dev_Cell_Agent_max_age[cell_id] = max_age;
  dev_Cell_Agent_sugar[cell_id] = endowment;
  dev_Cell_Agent_endowment[cell_id] = endowment;
  dev_Cell_Agent_metabolism[cell_id] = metabolism;
  dev_Cell_Agent_permission[cell_id] = false;

  hiprand_init(Cell_random_int(cell_id, 0, kSize*kSize), 0, 0,
  	          &dev_Cell_Agent_random_state[cell_id]);
}


__device__ void new_Male(int cell_id, int vision, int age, int max_age,
                         int endowment, int metabolism) {
  new_Agent(cell_id, vision, age, max_age, endowment, metabolism);
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;

  __threadfence();

  dev_Cell_Agent_type[cell_id] = kClassMale;
}


__device__ void new_Female(int cell_id, int vision, int age, int max_age,
                           int endowment, int metabolism, int max_children) {
  new_Agent(cell_id, vision, age, max_age, endowment, metabolism);
  dev_Cell_Female_num_children[cell_id] = 0;
  dev_Cell_Female_max_children[cell_id] = max_children;

  __threadfence();

  dev_Cell_Agent_type[cell_id] = kClassFemale;
}


__device__ void Agent_give_permission(int cell_id) {
  dev_Cell_Agent_permission[cell_id] = true;
}


__device__ void Agent_age_and_metabolize(int cell_id) {
  bool dead = false;

  dev_Cell_Agent_age[cell_id] = dev_Cell_Agent_age[cell_id] + 1;
  dead = dev_Cell_Agent_age[cell_id] > dev_Cell_Agent_max_age[cell_id];

  dev_Cell_Agent_sugar[cell_id] -= dev_Cell_Agent_metabolism[cell_id];
  dead = dead || dev_Cell_Agent_sugar[cell_id] <= 0;

  if (dead) {
    Cell_leave(cell_id);
    // No delete in baseline implementation.
  }
}


__device__ void Agent_prepare_move(int cell_id) {
  // Move to cell with the most sugar.
  int turn = 0;
  int target_cell = kNullptr;
  int target_sugar = 0;

  int this_x = cell_id % kSize;
  int this_y = cell_id / kSize;

  for (int dx = -dev_Cell_Agent_vision[cell_id];
       dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
    for (int dy = -dev_Cell_Agent_vision[cell_id];
         dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
      int nx = this_x + dx;
      int ny = this_y + dy;
      if ((dx != 0 || dy != 0)
          && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        int n_id = nx + ny*kSize;

        if (Cell_is_free(n_id)) {
          if (dev_Cell_sugar[n_id] > target_sugar) {
            target_cell = n_id;
            target_sugar = dev_Cell_sugar[n_id];
            turn = 1;
          } else if (dev_Cell_sugar[n_id] == target_sugar) {
            // Select cell with probability 1/turn.
            if (Agent_random_float(cell_id) <= 1.0f/(++turn)) {
              target_cell = n_id;
            }
          }
        }
      }
    }
  }

  dev_Cell_Agent_cell_request[cell_id] = target_cell;
}


__device__ void Agent_update_move(int cell_id) {
  if (dev_Cell_Agent_permission[cell_id]) {
    // Have permission to enter the cell.
    assert(dev_Cell_Agent_cell_request[cell_id] != kNullptr);
    assert(Cell_is_free(dev_Cell_Agent_cell_request[cell_id]));
    Cell_enter(dev_Cell_Agent_cell_request[cell_id], cell_id);
    Cell_leave(cell_id);
  }

  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
  dev_Cell_Agent_permission[cell_id] = false;
}


__device__ void Cell_prepare_diffuse(int cell_id) {
  dev_Cell_sugar_diffusion[cell_id] =
      kSugarDiffusionRate * dev_Cell_sugar[cell_id];
  int max_diff = kMaxSugarDiffusion;
  if (dev_Cell_sugar_diffusion[cell_id] > max_diff) {
    dev_Cell_sugar_diffusion[cell_id] = max_diff;
  }

  dev_Cell_sugar[cell_id] -= dev_Cell_sugar_diffusion[cell_id];
}


__device__ void Cell_update_diffuse(int cell_id) {
  int new_sugar = 0;
  int this_x = cell_id % kSize;
  int this_y = cell_id / kSize;

  for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
    for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
      int nx = this_x + dx;
      int ny = this_y + dy;
        if ((dx != 0 || dy != 0)
            && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        int n_id = nx + ny*kSize;

        // Add sugar from neighboring 8 cells.
        new_sugar += 0.125f * dev_Cell_sugar_diffusion[n_id];
      }
    }
  }

  dev_Cell_sugar[cell_id] += new_sugar;
}


__device__ void Cell_decide_permission(int cell_id) {
  int selected_agent = kNullptr;
  int turn = 0;
  int this_x = cell_id % kSize;
  int this_y = cell_id / kSize;

  for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
    for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
      int nx = this_x + dx;
      int ny = this_y + dy;
      if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        int n_id = nx + ny*kSize;

        if (dev_Cell_Agent_type[n_id] != kNoType
            && dev_Cell_Agent_cell_request[n_id] == cell_id) {
          ++turn;

          // Select cell with probability 1/turn.
          if (Cell_random_float(cell_id) <= 1.0f/turn) {
            selected_agent = n_id;
          } else {
            assert(turn > 1);
          }
        }
      }
    }
  }

  assert((turn == 0) == (selected_agent == kNullptr));

  if (selected_agent != kNullptr) {
    Agent_give_permission(selected_agent);
  }
}


__device__ void Cell_grow_sugar(int cell_id) {
  dev_Cell_sugar[cell_id] += min(
      dev_Cell_sugar_capacity[cell_id] - dev_Cell_sugar[cell_id],
      dev_Cell_grow_rate[cell_id]);
}


__device__ void Male_propose(int cell_id) {
  if (Agent_ready_to_mate(cell_id)) {
    // Propose to female with highest endowment.
    int target_agent = kNullptr;
    int target_sugar = -1;

    int this_x = cell_id % kSize;
    int this_y = cell_id / kSize;

    for (int dx = -dev_Cell_Agent_vision[cell_id];
         dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
      for (int dy = -dev_Cell_Agent_vision[cell_id];
           dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
        int nx = this_x + dx;
        int ny = this_y + dy;
        if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
          int n_id = nx + ny*kSize;

          if (dev_Cell_Agent_type[n_id] == kClassFemale
              && Agent_ready_to_mate(n_id)) {
            if (dev_Cell_Agent_sugar[n_id] > target_sugar) {
              target_agent = n_id;
              target_sugar = dev_Cell_Agent_sugar[n_id];
            }
          }
        }
      }
    }

    assert((target_sugar == -1) == (target_agent == kNullptr));
    dev_Cell_Male_female_request[cell_id] = target_agent;
  }
}


__device__ void Male_propose_offspring_target(int cell_id) {
  if (dev_Cell_Male_proposal_accepted[cell_id]) {
    assert(dev_Cell_Male_female_request[cell_id] != kNullptr);

    // Select a random cell.
    int target_cell = kNullptr;
    int turn = 0;

    int this_x = cell_id % kSize;
    int this_y = cell_id / kSize;

    for (int dx = -dev_Cell_Agent_vision[cell_id];
         dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
      for (int dy = -dev_Cell_Agent_vision[cell_id];
           dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
        int nx = this_x + dx;
        int ny = this_y + dy;
        if ((dx != 0 || dy != 0)
            && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
          int n_id = nx + ny*kSize;

          if (Cell_is_free(n_id)) {
            ++turn;

            // Select cell with probability 1/turn.
            if (Agent_random_float(cell_id) <= 1.0f/turn) {
              target_cell = n_id;
            }
          }
        }
      }
    }

    assert((turn == 0) == (target_cell == kNullptr));
    dev_Cell_Agent_cell_request[cell_id] = target_cell;
  }
}


__device__ void Male_mate(int cell_id) {
  if (dev_Cell_Male_proposal_accepted[cell_id]
      && dev_Cell_Agent_permission[cell_id]) {
    assert(dev_Cell_Male_female_request[cell_id] != kNullptr);
    assert(dev_Cell_Agent_cell_request[cell_id] != kNullptr);

    // Register birth.
    ++dev_Cell_Female_num_children[dev_Cell_Male_female_request[cell_id]];

    // Take sugar from endowment.
    int c_endowment = (dev_Cell_Agent_endowment[cell_id]
        + dev_Cell_Agent_endowment[dev_Cell_Male_female_request[cell_id]]) / 2;
    dev_Cell_Agent_sugar[cell_id] -= dev_Cell_Agent_endowment[cell_id] / 2;
    dev_Cell_Agent_sugar[dev_Cell_Male_female_request[cell_id]]
        -= dev_Cell_Agent_endowment[dev_Cell_Male_female_request[cell_id]] / 2;

    // Calculate other properties.
    int c_vision = (dev_Cell_Agent_vision[cell_id]
        + dev_Cell_Agent_vision[dev_Cell_Male_female_request[cell_id]]) / 2;
    int c_max_age = (dev_Cell_Agent_max_age[cell_id]
        + dev_Cell_Agent_max_age[dev_Cell_Male_female_request[cell_id]]) / 2;
    int c_metabolism = (dev_Cell_Agent_metabolism[cell_id]
        + dev_Cell_Agent_metabolism[dev_Cell_Male_female_request[cell_id]]) / 2;
    int c_max_children =
        dev_Cell_Female_max_children[dev_Cell_Male_female_request[cell_id]];

    // Create agent.
    assert(dev_Cell_Agent_cell_request[cell_id] != kNullptr);
    assert(dev_Cell_Agent_type[dev_Cell_Agent_cell_request[cell_id]] == kNoType);

    if (Agent_random_float(cell_id) <= 0.5f) {
      new_Male(dev_Cell_Agent_cell_request[cell_id],
               c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    } else {
      new_Female(dev_Cell_Agent_cell_request[cell_id],
                 c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism,
                 c_max_children);
    }

    // No Cell::enter necessary.
  }

  dev_Cell_Agent_permission[cell_id] = false;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
}


__device__ void Female_decide_proposal(int cell_id) {
  if (dev_Cell_Female_num_children[cell_id]
      < dev_Cell_Female_max_children[cell_id]) {
    int selected_agent = kNullptr;
    int selected_sugar = -1;
    int this_x = cell_id % kSize;
    int this_y = cell_id / kSize;

    for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
      for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
        int nx = this_x + dx;
        int ny = this_y + dy;
        if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
          int n_id = nx + ny*kSize;

          if (dev_Cell_Agent_type[n_id] == kClassMale) {
            if (dev_Cell_Male_female_request[n_id] == cell_id
                && dev_Cell_Agent_sugar[n_id] > selected_sugar) {
              selected_agent = n_id;
              selected_sugar = dev_Cell_Agent_sugar[n_id];
            }
          }
        }
      }
    }

    assert((selected_sugar == -1) == (selected_agent == kNullptr));

    if (selected_agent != kNullptr) {
      dev_Cell_Male_proposal_accepted[selected_agent] = true;
    }
  }
}


__global__ void kernel_Cell_grow_sugar() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_grow_sugar(i);
  }
}


__global__ void kernel_Cell_prepare_diffuse() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_prepare_diffuse(i);
  }
}


__global__ void kernel_Cell_update_diffuse() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_update_diffuse(i);
  }
}


__global__ void kernel_Agent_age_and_metabolize() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_age_and_metabolize(i);
    }
  }
}


__global__ void kernel_Agent_prepare_move() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_prepare_move(i);
    }
  }
}


__global__ void kernel_Cell_decide_permission() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_decide_permission(i);
  }
}


__global__ void kernel_Agent_update_move() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_update_move(i);
    }
  }
}


__global__ void kernel_Agent_harvest_sugar() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      // Must be in a separate kernel to avoid race condition.
      // (Old and new cell could both be processed.)
      Agent_harvest_sugar(i);
    }
  }
}


__global__ void kernel_Male_propose() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_propose(i);
    }
  }
}


__global__ void kernel_Female_decide_proposal() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassFemale) {
      Female_decide_proposal(i);
    }
  }
}


__global__ void kernel_Male_propose_offspring_target() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_propose_offspring_target(i);
    }
  }
}


__global__ void kernel_Male_mate() {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_mate(i);
    }
  }
}


void step() {
  kernel_Cell_grow_sugar<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_prepare_diffuse<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_update_diffuse<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_age_and_metabolize<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_prepare_move<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_decide_permission<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_update_move<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_harvest_sugar<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Male_propose<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Female_decide_proposal<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Male_propose_offspring_target<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_decide_permission<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Male_mate<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
}


__global__ void create_cells() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    new_Cell(i, kSeed, /*sugar=*/ 0, kSugarCapacity, /*max_grow_rate=*/ 50);
  }
}


__global__ void create_agents() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    float r = Cell_random_float(i);
    int c_vision = kMaxVision/2 + Cell_random_int(i, 0, kMaxVision/2);
    int c_max_age = kMaxAge*2/3 + Cell_random_int(i, 0, kMaxAge/3);
    int c_endowment = kMaxEndowment/4
                      + Cell_random_int(i, 0, kMaxEndowment*3/4);
    int c_metabolism = kMaxMetabolism/3
                       + Cell_random_int(i, 0, kMaxMetabolism*2/3);
    int c_max_children = Cell_random_int(i, 2, kMaxChildren);

    if (r < kProbMale) {
      // Create male agent.
      new_Male(i, c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    } else if (r < kProbMale + kProbFemale) {
      // Create female agent.
      new_Female(i, c_vision, /*age=*/ 0, c_max_age, c_endowment,
                 c_metabolism, c_max_children);
    }   // else: Do not create agent.
  }
}


void initialize_simulation() {
  create_cells<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  create_agents<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


int data_Cell_sugar[kSize*kSize];
char data_Cell_agent_types[kSize*kSize];
int checksum(int* host_Cell_sugar, char* host_Cell_Agent_type) {
  hipMemcpy(data_Cell_sugar, host_Cell_sugar, sizeof(int)*kSize*kSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(data_Cell_agent_types, host_Cell_Agent_type,
             sizeof(char)*kSize*kSize, hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  int result = 0;
  for (int i = 0; i < kSize*kSize; ++i) {
    result += data_Cell_agent_types[i]; //(data_Cell_sugar[i] * i) % 1234567;
  }
  return result;
}


int main(int /*argc*/, char** /*argv*/) {
  // Allocate memory.
  hiprandState_t* host_Cell_random_state;
  hipMalloc(&host_Cell_random_state, sizeof(hiprandState_t)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_random_state), &host_Cell_random_state,
                     sizeof(hiprandState_t*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar_diffusion;
  hipMalloc(&host_Cell_sugar_diffusion, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar_diffusion), &host_Cell_sugar_diffusion,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar;
  hipMalloc(&host_Cell_sugar, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar), &host_Cell_sugar,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar_capacity;
  hipMalloc(&host_Cell_sugar_capacity, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar_capacity), &host_Cell_sugar_capacity,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_grow_rate;
  hipMalloc(&host_Cell_grow_rate, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_grow_rate), &host_Cell_grow_rate,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  char* host_Cell_Agent_type;
  hipMalloc(&host_Cell_Agent_type, sizeof(char)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_type), &host_Cell_Agent_type,
                     sizeof(char*), 0, hipMemcpyHostToDevice);

  hiprandState_t* host_Cell_Agent_random_state;
  hipMalloc(&host_Cell_Agent_random_state, sizeof(hiprandState_t)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_random_state), &host_Cell_Agent_random_state,
                     sizeof(hiprandState_t*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_cell_request;
  hipMalloc(&host_Cell_Agent_cell_request, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_cell_request), &host_Cell_Agent_cell_request,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_vision;
  hipMalloc(&host_Cell_Agent_vision, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_vision), &host_Cell_Agent_vision,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_age;
  hipMalloc(&host_Cell_Agent_age, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_age), &host_Cell_Agent_age,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_max_age;
  hipMalloc(&host_Cell_Agent_max_age, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_max_age), &host_Cell_Agent_max_age,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_sugar;
  hipMalloc(&host_Cell_Agent_sugar, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_sugar), &host_Cell_Agent_sugar,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_metabolism;
  hipMalloc(&host_Cell_Agent_metabolism, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_metabolism), &host_Cell_Agent_metabolism,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_endowment;
  hipMalloc(&host_Cell_Agent_endowment, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_endowment), &host_Cell_Agent_endowment,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  bool* host_Cell_Agent_permission;
  hipMalloc(&host_Cell_Agent_permission, sizeof(bool)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_permission), &host_Cell_Agent_permission,
                     sizeof(bool*), 0, hipMemcpyHostToDevice);

  int* host_Cell_female_num_children;
  hipMalloc(&host_Cell_female_num_children, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Female_num_children), &host_Cell_female_num_children,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_female_max_children;
  hipMalloc(&host_Cell_female_max_children, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Female_max_children), &host_Cell_female_max_children,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Male_female_request;
  hipMalloc(&host_Cell_Male_female_request, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Male_female_request),
                     &host_Cell_Male_female_request,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  bool* host_Cell_Male_proposal_accepted;
  hipMalloc(&host_Cell_Male_proposal_accepted, sizeof(bool)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Male_proposal_accepted),
                     &host_Cell_Male_proposal_accepted,
                     sizeof(bool*), 0, hipMemcpyHostToDevice);
  assert(host_Cell_Male_proposal_accepted != nullptr);

  gpuErrchk(hipDeviceSynchronize());

  initialize_simulation();

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumIterations; ++i) {
    step();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto micros = std::chrono::duration_cast<std::chrono::microseconds>(elapsed)
      .count();

#ifndef NDEBUG
  printf("Checksum: %i\n", checksum(host_Cell_sugar, host_Cell_Agent_type));
#endif  // NDEBUG

  printf("%lu\n", micros);

  return 0;

  // TODO: Free CUDA memory.
}
