#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#include "linux_scalability.h"

// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;


__global__ void  kernel_benchmark(int num_alloc, DummyClass** ptrs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  DummyClass** my_ptrs = ptrs + tid*num_alloc;

  for (int i = 0; i < num_alloc; ++i) {
    DummyClass* p = new(device_allocator) DummyClass();
    my_ptrs[i] = p;
    if (p == nullptr) {
      asm("trap;");
    }
  }

  for (int i = 0; i < num_alloc; ++i) {
    destroy(device_allocator, my_ptrs[i]);
  }
}


int main() {
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  DummyClass** ptrs;
  hipMalloc(
      &ptrs, sizeof(DummyClass*)*kNumBlocks*kNumThreads*kNumAllocPerThread);
  gpuErrchk(hipDeviceSynchronize());

  auto time_before = std::chrono::system_clock::now();

  // Run benchmark.
  // TODO: Will run OOM with custom allocators if >1 iterations. (ptr array)
  for (int i = 0; i < kNumIterations; ++i) {
    kernel_benchmark<<<kNumBlocks, kNumThreads>>>(kNumAllocPerThread, ptrs);
    gpuErrchk(hipDeviceSynchronize());
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_before;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("%lu,%lu\n", millis, allocator_handle->DBG_get_enumeration_time());
}
