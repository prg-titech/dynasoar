#include "hip/hip_runtime.h"
#include "dynasoar.h"

#ifdef PARAM_HEAP_SIZE
static const int kHeapSize = PARAM_HEAP_SIZE;
#else
static const int kHeapSize = 64*64*64;
#endif  // PARAM_HEAP_SIZE


// Pre-declare all classes.
class Foo;
class Bar;


// Declare allocator type. First argument is max. number of objects that can be created.
using AllocatorT = SoaAllocator<kHeapSize, Bar, Foo>;


// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;

#if GCC_COMPILER
class Bar : public AllocatorT::Base {
#else
// Workaround for compiler other than GCC. Will fix this in future versions.
class Bar : public SoaBase<AllocatorT> {
#endif  // GCC_COMPILER
 public:
  // Pre-declare types of all fields.
  declare_field_types(Bar, Foo*, int)

  __device__ Bar(Foo* f0, int f1) : field0_(f0), field1_(f1) {}

  Field<Bar, 0> field0_;
  Field<Bar, 1> field1_;

  __device__ void foo(int v);

  __device__ void assert_result() {
    if (field1_ != 1 + 2 + 3 + 4 + 5) {
      printf("Incorrect result!\n");
      asm("trap;");  // Force kernel to quit.
    }
  }
};


#if GCC_COMPILER
class Foo : public AllocatorT::Base {
#else
// Workaround for compiler other than GCC. Will fix this in future versions.
class Foo : public SoaBase<AllocatorT> {
#endif  // GCC_COMPILER
 public:
  // Pre-declare types of all fields.
  declare_field_types(Foo, int, int, int)
  
  // Declare fields.
  SoaField<Foo, 0> field0_;  // int
  SoaField<Foo, 1> field1_;  // int
  SoaField<Foo, 2> field2_;  // int
  
  __device__ Foo(int f0, int f1, int f2)
      : field0_(f0), field1_(f1), field2_(f2) {}
 
  __device__ int qux() {
    return field0_ + field1_ + field2_;
  }
};


__device__ void Bar::foo(int v) {
  field1_ += field0_->qux() + v;
}


__global__ void create_objects() {
  auto* f = new(device_allocator) Foo(1, 2, 3);
  // Delete objects with: destroy(device_allocator, ptr)

  new(device_allocator) Bar(f, 4);
}


int main(int /*argc*/, char** /*argv*/) {
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Create 2048 objects of Foo and Bar.
  create_objects<<<32, 64>>>();
  hipDeviceSynchronize();

  // Call Bar::foo on all 2048 objects.
  allocator_handle->parallel_do<Bar, int, &Bar::foo>(5);

  // Check correctness.
  allocator_handle->parallel_do<Bar, &Bar::assert_result>();

  printf("Check passed!\n");
  return 0;
}
